#include <iostream>
#include <hip/hip_runtime.h>

#include "/nndescent/GPU_HDBSCAN/experiments/tools/filetool.hpp"
#include "structs/hdbscan_elements.cuh"
#include "structs/ECLgraph.h"
#include "graphs/graph.cuh"

#include <climits>
#include <algorithm>
#include <tuple>
#include <vector>
#include <sys/time.h>
#include "../structs/ECLgraph.h"


static const int Device = 0;
static const int ThreadsPerBlock = 512;

typedef unsigned long long ull;


void generate_random(int *h_data){

   // Gera o vetor de teste
  for (long int i = 0; i < vectorSize; i++) {
        h_data[i] = /*i / k;*/rand() % numValues;
    }   

    return;
}


void generate_random(float *h_data){

   // Gera o vetor de teste
  for (long int i = 0; i < vectorSize; i++) {
        h_data[i] = /*i / k;*/rand() % 50 + 0.15 * (rand() % 50);
    }   

    return;
}






int main() {


    int shards_num = 3;

    // Le o kNNG que esta escrito no arquivo abaixo
    std::string path_to_kNNG = "/nndescent/GPU_HDBSCAN/experiments/results/NNDescent-KNNG.kgraph";
    NNDElement *result_graph;
    int num, dim;
    FileTool::ReadBinaryVecs(path_to_kNNG , &result_graph, &num, &dim);
    num = numValues;
    printf("%d e %d\n",num,dim);




    int *result_index_graph;
    hipMallocManaged(&result_index_graph,(size_t)num*dim * sizeof(int));
    for (long int i = 0; i < num; i++) {
      for (long int j = 0; j < dim; j++) {

        result_index_graph[i * dim + j] = result_graph[i * dim + j].label();
      }

    } 



    CheckCUDA_();

   

    float *distances;
    hipMallocManaged(&distances,(size_t)numValues*dim * sizeof(float));


    for (long int i = 0; i < num; i++) {
        for (long int j = 0; j < dim; j++) {
          
          distances[i * dim + j] = result_graph[i * dim + j].distance();
        }
    }

    CheckCUDA_();



    ECLgraph g;
    g = buildEnhancedKNNG(result_index_graph,distances,shards_num);

    printf("O grafo tem %d NOHS e %ld arestas\n",g.nodes,g.edges);
     bool* edges = cpuMST(&g);

     



  return 0;


}
