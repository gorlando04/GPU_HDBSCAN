#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>

#include "/nndescent/GPU_HDBSCAN/experiments/tools/filetool.hpp"
#include "structs/hdbscan_elements.cuh"
#include "structs/ECLgraph.h"
#include "graphs/graph.cuh"
#include "mst/mst.cuh"
#include "trees/tree.cuh"

#include <fstream>
#include <string>
#include <cstdio>


void generate_random(int *h_data){

   // Gera o vetor de teste
  for (long int i = 0; i < vectorSize; i++) {
        h_data[i] = /*i / k;*/rand() % numValues;
    }   

    return;
}


void generate_random(float *h_data){

   // Gera o vetor de teste
  for (long int i = 0; i < vectorSize; i++) {
        h_data[i] = /*i / k;*/rand() % 50 + 0.15 * (rand() % 50);
    }   

    return;
}

void ReadTxtVecs(const string &data_path, float **vectors_ptr,
                          long int *num_ptr, long int *dim_ptr,
                          const bool show_process = true) {
    float *&vecs = *vectors_ptr;
    long int &num = *num_ptr;
    long int &dim = *dim_ptr;
    std::ifstream in(data_path);
    if (!in.is_open()) {
      throw(std::string("Failed to open ") + data_path);
    }
    in >> num >> dim;
    std::cerr << num << " " << dim << std::endl;
    vecs = new float[num * dim];
   printf("%ld deu bom?\n",num*dim);

   for (int i = 0; i < num; i++) {

      for (int j = 0; j < dim; j++) {
        in >> vecs[i * dim + j];
      }
    }

    in.close();
    delete in;
    return;
  }

void WriteTxtVecs(const string &data_path, const int *vectors,
                           const int write_num) {
    ofstream out(data_path);

    for (int i = 0; i < write_num; i++) {
      out << write_num << '\n';
        out << vectors[i] << '\t';
      out << endl;
    }
    out.close();
    return;
}






int main() {


    int shards_num = 3;

     clock_t t; 
    t = clock(); 
    // Le o kNNG que esta escrito no arquivo abaixo
    std::string path_to_kNNG = "/nndescent/GPU_HDBSCAN/experiments/results/NNDescent-KNNG.kgraph";
    NNDElement *result_graph;
    int num, dim;
    FileTool::ReadBinaryVecs(path_to_kNNG , &result_graph, &num, &dim);
    num = numValues;
    printf("kNNG size = %ld e %d\n",num,dim);
    
    // Le o vetor de amostras
    float *vectors_data;
    int vecs_size, dim_;
    const std::string path_to_data = "/nndescent/GPU_KNNG/data/artificial/SK_data.txt";

    ReadTxtVecs(path_to_data,&vectors_data,&vecs_size,&dim_);
    printf("Data size= %d e %d\n",numValues,dim_);




    int *result_index_graph;
    hipMallocManaged(&result_index_graph,(size_t)num*dim * sizeof(int));
    for (long int i = 0; i < num; i++) {
      for (long int j = 0; j < dim; j++) {

        result_index_graph[i * dim + j] = result_graph[i * dim + j].label();
      }

    } 

    CheckCUDA_();

   

    float *distances;
    hipMallocManaged(&distances,(size_t)numValues*dim * sizeof(float));


    for (long int i = 0; i < num; i++) {
        for (long int j = 0; j < dim; j++) {
          
          distances[i * dim + j] = result_graph[i * dim + j].distance();
        }
    }

    CheckCUDA_();



    ECLgraph g;
    g = buildEnhancedKNNG(result_index_graph,distances,shards_num,vectors_data,dim_);

    printf("O grafo tem %d NOHS e %ld arestas\n",g.nodes,g.edges);
     bool* edges = cpuMST(g);

     
    t = clock() - t; 
    double time_taken = ((double)t)/CLOCKS_PER_SEC; // in seconds 
    printf("Demorou %lf segundos para tudo\n",time_taken);

     MSTedge *mst_edges;
     mst_edges = new MSTedge[g.nodes-1];

     mst_edges = buildMST(g,edges,12);

    SingleLinkageNode *result_arr;

    result_arr = build_Linkage_tree(mst_edges ,num ,g.nodes);

    CondensedTreeNode* condensed_tree;
    int condensed_size;
    condensed_tree =  build_Condensed_tree(result_arr, num ,g.nodes-1, k,&condensed_size);

    for(int i=0;i<condensed_size;i++){

      if (condensed_tree[i].parent == 1000120)
        printf("PARENT: %d CHILD: %d SIZE: %d\n",condensed_tree[i].parent,condensed_tree[i].child, condensed_tree[i].child_size);
    }




    Stability *stabilities;
    int stability_size;
    
    stabilities = compute_stability(condensed_tree,condensed_size,&stability_size);

    int* labels;
    labels = get_clusters(condensed_tree, condensed_size, stabilities,  stability_size, numValues);


    const std::string out_PATH = "/nndescent/GPU_HDBSCAN/HDBSCAN/groudtruth/approximate_result.txt";
    WriteTxtVecs(out_PATH,labels,100);



  return 0;


}
