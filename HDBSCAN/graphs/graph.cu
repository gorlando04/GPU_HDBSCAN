#include "hip/hip_runtime.h"
#include "graph.cuh"
#include "../initializer/initialize.cuh"
#include "../getters/getters.cuh"
#include "../calculates/calculates.cuh"
#include "../counts/count.cuh"
#include <time.h>
#include <unistd.h>
#include "../../tools/filetool.hpp"

#include <algorithm>
#include <vector>
#include <omp.h>
#include <pthread.h>


void joinAntiHubs(int *antihubs,Vertex *vertexes,int not_ties, Untie_hub *unties,int missing_ties){

    // Bota os não empatados
    for(int i=0;i< not_ties;i++){
        antihubs[i] = vertexes[i].index;

    }

    for(int i=0;i<missing_ties;i++){
        antihubs[i+not_ties] = unties[i].index;
    }

    return ;
}





void createNodeList(int *vector,ECLgraph *g){

    for(int i=0;i<g->nodes;i++){
        
        long int begin = g->nindex[i];
        long int end = g->nindex[i+1];

        for (long int j=begin;j<end;j++)
            vector[j] = i;
    }
}

void createNodeList_gpu(int *vector,GPUECLgraph *g){

    for(int i=0;i<g->nodes;i++){

        long int begin = g->nindex[i];
        long int end = g->nindex[i+1];

        for (long int j=begin;j<end;j++)
            vector[j] = i;
    }
}



void createEdgeList(int *vector,ECLgraph *g){

    for(long int i=0;i<g->edges;i++){
        
        vector[i] = g->nlist[i];
    }
}

void createWeightList(float *vector,ECLgraph *g){

    for(long int i=0;i<g->edges;i++){
        
        vector[i] = g->eweight[i];
    }
}

ECLgraph buildECLgraph(int nodes, long int edges,int *kNN, float *distances,int k,long int mpts, int *antihubs, long int num_antihubs,float *vectors_data,int dim, long int numValues,int num_buckets,int num_threads  ,int mst_gpu)
{



  ECLgraph g;


  g.nodes = nodes;



   /*Isso significa, que o nó 0 está conectado com Y-x NÓS,
   O nó 1 está conetado com z-y nós, e etc...*/
    hipMallocManaged(&g.nindex,(size_t)(g.nodes + 1) * sizeof(g.nindex[0])); // nindex[0] = X, nindex[1] = y, nindex[2] = z

    int gridSize = (g.nodes + 1 + blockSize - 1) / blockSize;

    
    initializeVectorCounts_<<<gridSize,blockSize>>>(g.nindex,0,g.nodes+1); // Aqui usar GPU

    hipDeviceSynchronize();

    CheckCUDA_();
    printf("Vetores inicializados com sucesso - 1\n");
    hipMemPrefetchAsync(g.nindex,(size_t)(g.nodes + 1) * sizeof(g.nindex[0]),hipCpuDeviceId);
    bool *flag_knn = (bool*)malloc(numValues*k * sizeof(bool));


    // Calcula quantas arestas cada noh terá, levando em conta que eh um grafo não direcional.
    for (long int i=0;i<nodes;i++){
        long int soma = 0;
        bool temp=false;
        for (long int j=0;j<k;j++){

            long int neig = kNN[i*k + j];

            //Verifica se i esta na lista de neig
            int FLAG = findKNNlist(kNN,neig,i,k);
	    flag_knn[i*k + j] = FLAG;
            if (FLAG > 1){ g.nindex[neig+1] += FLAG-1; g.nindex[i+1] -= (FLAG-1);}

            g.nindex[neig+1] += 1;
           
            if (!FLAG)
                soma += 1;
        }
        g.nindex[i+1] += soma;
    }

    // Adicionar os antihubs
    int contador = 0;

    for (long int i=0;i<nodes;i++)

        if (i == antihubs[contador]){
            contador ++;
            g.nindex[i+1] += (num_antihubs-1);
    }
     
    
    //Calcular offsets
    for (long int i=1;i<nodes+1;i++){

        g.nindex[i] = g.nindex[i-1] + g.nindex[i];

    }


    // Nesse pontos os nós já estão calculados, agora precisamos inserir as arestas. Essa parte será bem demorada.

    long int *auxiliar_edges;

    hipMallocManaged(&auxiliar_edges,(size_t)(g.nodes) * sizeof(long int)); // nindex[0] = X, nindex[1] = y, nindex[2] = z

    gridSize = (g.nodes + 1  + blockSize - 1) / blockSize;


    initializeVectorCounts_<<<gridSize,blockSize>>>(auxiliar_edges,0,g.nodes); //Aqui usar GPU
    hipMemPrefetchAsync(auxiliar_edges,(size_t)(g.nodes ) * sizeof(long int),hipCpuDeviceId); //Inserir no código

    hipDeviceSynchronize();

    CheckCUDA_();
    printf("Vetores inicializados com sucesso - 2\n");


    hipMallocManaged(&g.nlist,(size_t)(g.nindex[nodes]) * sizeof(int));
    g.edges = g.nindex[nodes];

    // Adicionar as arestas sem antihubs
    long int k2 = k;

/*     omp_set_num_threads(32);
    #pragma omp parallel for 
    for (long int i = 0; i < nodes; i++) {
        
        // Calcula o offset do ponto
        long int edge_offset = g.nindex[i];
        long int pos = edge_offset + auxiliar_edges[i];    

        for (long int j = 0; j < k2; j++) {
        
            // Pega o   ndice do vizinho  
            long int neig = kNN[i * k2 + j];

            g.nlist[pos] = neig;
            auxiliar_edges[i] += 1;

            pos += 1; 

        }
    }


*/
 clock_t t; 
    t = clock();

  /*  pthread_mutex_t mutexes[num_buckets];

    for(int i=0;i<num_buckets;i++)
        pthread_mutex_init(&mutexes[i],NULL);
    

    omp_set_num_threads(num_threads);
    #pragma omp parallel for 
    for (long int i = 0; i < nodes; i++) {
        

        for (long int j = 0; j < k2; j++) {
        
            // Pega o   indice do vizinho  
            long int neig = kNN[i * k2 + j];

            int FLAG = flag_knn[i*k2+j]; 

            // Deu problema
            if (!FLAG){

                //Calcula Propriedades de NEIG em NList
                long int neig_edge_offset = g.nindex[neig];

                int bucket = neig % num_buckets;

                pthread_mutex_lock(&mutexes[bucket]);

                long int neig_pos = neig_edge_offset + auxiliar_edges[neig];
                // Adicionando o idx i na lista do neig
                auxiliar_edges[neig] += 1;
                g.nlist[neig_pos] = i;

                pthread_mutex_unlock(&mutexes[bucket]);


            }
        }
    }

    for(int i=0;i<num_buckets;i++)
        pthread_mutex_destroy(&mutexes[i]);
*/

     omp_set_num_threads(32);
    #pragma omp parallel for 
    for (long int i = 0; i < nodes; i++) {
        
        // Calcula o offset do ponto
        long int edge_offset = g.nindex[i];
        long int pos = edge_offset + auxiliar_edges[i];    

        for (long int j = 0; j < k2; j++) {
        
            // Pega o   ndice do vizinho  
            long int neig = kNN[i * k2 + j];

            g.nlist[pos] = neig;
            auxiliar_edges[i] += 1;

            pos += 1; 

        }
    }


    for (long int i = 0; i < nodes; i++) {
        

        for (long int j = 0; j < k2; j++) {
        
            // Pega o   indice do vizinho  
            long int neig = kNN[i * k2 + j];

            int FLAG = flag_knn[i*k2+j]; 

            // Deu problema
            if (!FLAG){

                //Calcula Propriedades de NEIG em NList
                long int neig_edge_offset = g.nindex[neig];


                long int neig_pos = neig_edge_offset + auxiliar_edges[neig];
                // Adicionando o idx i na lista do neig
                auxiliar_edges[neig] += 1;
                g.nlist[neig_pos] = i;


            }
        }
    }


    t = clock() - t; 
    double time_taken = ((double)t)/CLOCKS_PER_SEC; // in seconds */

    printf("Demorou %lf segundos com BUCKETS = %d e THREADS = %d\n",time_taken,num_buckets,num_threads);


    for (long int i=0;i<num_antihubs;i++){

        int current = antihubs[i];
        long int pos_begin = g.nindex[current];
        long int offset = auxiliar_edges[current];

        for (long int j=0;j<num_antihubs;j++){
            int neig = antihubs[j];

            if (neig != current){
                g.nlist[pos_begin+offset] = neig;
                offset++;
            }
        }
    }


        hipMallocManaged(&g.eweight,(size_t)g.edges * sizeof(g.eweight[0]));

	 int *aux_nodes;
        hipMallocManaged(&aux_nodes,(size_t)g.edges * sizeof(g.nlist[0]));
        createNodeList(aux_nodes,&g) ; //Aqui usar GPU

    hipDeviceSynchronize();

    CheckCUDA_();
printf("Vetores inicializados com sucesso - 3\n");
         
        long int elementsPerGPU[numGPUs];
        calculateElements(elementsPerGPU,numGPUs,numValues); 

        float *coreDistances;
        hipMallocManaged(&coreDistances,(size_t)(numValues) * sizeof(float)); 
        calculateCoreDistance(distances,coreDistances,elementsPerGPU,k,mpts); //Aqui usa GPU

    hipDeviceSynchronize();

    CheckCUDA_();
printf("Vetores inicializados com sucesso - 4 - %f\n",coreDistances[numValues-1]);

    hipFree(kNN);
    
    kNN = NULL;

    if(mst_gpu != 1){
	hipFree(distances); distances = NULL; }


    calculateMutualReachabilityDistance(g.eweight,coreDistances,aux_nodes,g.nlist,g.edges);  //Aqui usa GPU

    hipDeviceSynchronize();

    CheckCUDA_();

printf("Vetores inicializados com sucesso - 5\n");


    // Read vector txtx
    const std::string path_to_data = "/nndescent/GPU_HDBSCAN/data/artificial/SK_data.txt";
    const std::string path_to_data_binary = "/nndescent/GPU_HDBSCAN/data/vectors.fvecs";
    long int data_size2, data_dim2;

    FileTool::ReadBinaryAntihubs(path_to_data_binary,&vectors_data, &data_size2, &data_dim2,antihubs,num_antihubs);    	

    for (long int i=0;i<num_antihubs;i++){

        int idx_a = antihubs[i];
        long int pos_begin = g.nindex[idx_a] + auxiliar_edges[idx_a];
        for (long int j=i;j<num_antihubs-1;j++){
            int idx_b = antihubs[j+1];
            
            //Calcula distancia euclidiana
            float euclidean_distance = calculate_euclidean_distance(vectors_data,i,j+1,dim);

            if (g.eweight[pos_begin + j] < euclidean_distance){
                g.eweight[pos_begin+j] = euclidean_distance;

                long int pos_begin2 = g.nindex[idx_b] + auxiliar_edges[idx_b];
                g.eweight[pos_begin2 + i] = euclidean_distance;
            }


        }
    }


  return g;   
}


ECLgraph buildEnhancedKNNG(int *h_data, float *distances, int shards_num,float *vectors_data,int dim, long int numValues,long int k,long int mpts,int num_buckets,int num_threads  ,int mst_gpu){




    long int vectorSize = numValues*k;

   hipMemPrefetchAsync(h_data,(size_t)vectorSize * sizeof(int), hipCpuDeviceId);
 
    long int elementsPerGPU[shards_num];

    // Calcula a quantidade de elementos por GPU
    calculateElements(elementsPerGPU,shards_num,vectorSize);


    // Realiza a contagem de graus para todos os vértices
    int *finalCounts; // Contagens finais após a combinação das GPUs

    hipMallocManaged(&finalCounts,(size_t)numValues * sizeof(int));

    
    int gridSize = (numValues + blockSize - 1) / blockSize;

    // Inicializa o vetor
    initializeVectorCounts<<<gridSize,blockSize>>>(finalCounts,0,numValues);

    hipDeviceSynchronize();

    auto cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        printf("%s hehehehehe",hipGetErrorString(cuda_status));
        exit(-1);
    }

    // Conta os graus de cada vértice
  countDegrees(finalCounts,h_data,numGPUs,elementsPerGPU,numValues);

    Vertex *vertexes;

    //vertexes = new Vertex[numValues];
    // Aloca memória para o vetor na CPU
	hipMallocManaged(&vertexes,(size_t)numValues * sizeof(Vertex));


  
    //Configura a grade de threads
    gridSize = (numValues + blockSize - 1) / blockSize;    
    
    // Inicializa o vetor de vértices, com  os graus específicos.
    initializeVertex<<<gridSize,blockSize>>>(vertexes,finalCounts,numValues);

    hipDeviceSynchronize();

CheckCUDA_();

    // Pegar os threshold
    int pos_threshold = get_NumThreshold(numValues);

    // Ordenação Parcial
    std::partial_sort(vertexes, vertexes + pos_threshold, vertexes + numValues, compareVertexByDegree);


    // Pegar o valor do threshold 
    int value_threshold = vertexes[pos_threshold-1].grau;

    printf("A posicao do threshold eh: %d e o valor eh: %d\n",pos_threshold-1,value_threshold);


    // Evita page fault
    for (int i=0;i<numGPUs;i++){
        hipSetDevice(i);
        hipMemPrefetchAsync(vertexes,(size_t)numValues * sizeof(Vertex),i);
    }

    // Encontrar quantos valores são iguais ao threshold
    long int elementsPerGPU_[numGPUs];
    calculateElements(elementsPerGPU_,numGPUs,numValues);


    // Encontra quantos valores são iguais ao threshold
    int countsTreshold = countThreshold_(elementsPerGPU_,vertexes,value_threshold);

    int *treshold_idx;

    int *antihubs;

    antihubs = new int[pos_threshold];

     Untie_hub *unties;

     if (countsTreshold > 1){
    // Aloca memória para o vetor na CPU
	hipMallocManaged(&treshold_idx,(size_t)countsTreshold * sizeof(int));
  


    CheckCUDA_();

    // Pega os índices dos pontos que são iguais ao threshold
    get_IndexThreshold(finalCounts,treshold_idx,value_threshold,numValues);



    for (int i=0;i<numGPUs;i++){
        hipSetDevice(i);
        hipMemPrefetchAsync(treshold_idx,(size_t)countsTreshold * sizeof(int),i);
    }

    long int indexesPerGPU[numGPUs];

    // Calcula quantos elementos serão processados por cada GPU
    calculateElements(indexesPerGPU,numGPUs,countsTreshold);


    unties = new Untie_hub[countsTreshold];

    // Calculata os scores dos empates
    calculateUntieScore(unties,indexesPerGPU,h_data,treshold_idx,finalCounts,k);

    // Pega quantos empates temos na lista final
    int missing_ties = get_TiedVertexes(vertexes,pos_threshold,value_threshold);
    int not_ties = pos_threshold - missing_ties;

    std::partial_sort(unties, unties + missing_ties, unties + countsTreshold, compareVertexByScore);

    // Junta todos os antihubs em um vetor
    joinAntiHubs(antihubs,vertexes,not_ties,unties,missing_ties);

    delete unties;
    unties = NULL;
    }

    else{
        // Bota os não empatados
        for(int i=0;i< pos_threshold;i++)
            antihubs[i] = vertexes[i].index;

    }
    // Ordena pelo índice para inserir na MST
    std::sort(antihubs,antihubs+pos_threshold);


    // Libera a galera
    hipFree(finalCounts);
    hipFree(vertexes);
    hipFree(treshold_idx);

    finalCounts = NULL;
    vertexes = NULL;
    treshold_idx = NULL;


    ECLgraph g;
    
    g = buildECLgraph(numValues, vectorSize,h_data, distances,k,mpts, antihubs, pos_threshold,vectors_data,dim,numValues,num_buckets,num_threads  ,mst_gpu);

    return g;
}
