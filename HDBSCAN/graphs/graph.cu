#include "graph.cuh"
#include "../initializer/initialize.cuh"
#include "../getters/getters.cuh"
#include "../calculates/calculates.cuh"
#include "../counts/count.cuh"
#include <time.h>
#include <unistd.h>
#include "../../tools/filetool.hpp"

#include <algorithm>
#include <vector>
#include <omp.h>
#include <pthread.h>


void joinAntiHubs(int *antihubs,Vertex *vertexes,int not_ties, Untie_hub *unties,int missing_ties){

    // Bota os não empatados
    for(int i=0;i< not_ties;i++){
        antihubs[i] = vertexes[i].index;

    }

    for(int i=0;i<missing_ties;i++){
        antihubs[i+not_ties] = unties[i].index;
    }

    return ;
}

void createNodeList(int *vector,ECLgraph *g){

    for(int i=0;i<g->nodes;i++){
        
        long int begin = g->nindex[i];
        long int end = g->nindex[i+1];

        for (long int j=begin;j<end;j++)
            vector[j] = i;
    }
}

void createNodeList_gpu(int *vector,GPUECLgraph *g){

    for(int i=0;i<g->nodes;i++){

        long int begin = g->nindex[i];
        long int end = g->nindex[i+1];

        for (long int j=begin;j<end;j++)
            vector[j] = i;
    }
}


void createEdgeList(int *vector,ECLgraph *g){

    for(long int i=0;i<g->edges;i++){
        
        vector[i] = g->nlist[i];
    }
}

void createWeightList(float *vector,ECLgraph *g){

    for(long int i=0;i<g->edges;i++){
        
        vector[i] = g->eweight[i];
    }
}

void Check(){

    hipDeviceSynchronize();

    CheckCUDA_();

    return;

}

template <typename T>
void avoid_pageFault(T *array,int numValues,int is_cpu=false){


    if (is_cpu)
        hipMemPrefetchAsync(array,(size_t)numValues * sizeof(T),hipCpuDeviceId);
        return;

    // Evita page fault
    for (int i=0;i<numGPUs;i++){
        hipSetDevice(i);
        hipMemPrefetchAsync(array,(size_t)numValues * sizeof(T),i);
    }
}

void calculate_nindex(int nodes, int *kNN, bool *flag_knn,ECLgraph *g,int *antihubs,int num_antihubs){

 // Calcula quantas arestas cada noh terá, levando em conta que eh um grafo não direcional.
    for (long int i=0;i<nodes;i++){

        long int soma = 0;


        for (long int j=0;j<k;j++){

            long int neig = kNN[i*k + j];

            //Verifica se i esta na lista de neig
            int FLAG = findKNNlist(kNN,neig,i,k);
	        flag_knn[i*k + j] = FLAG;

            if (FLAG > 1){ g->nindex[neig+1] += FLAG-1; g->nindex[i+1] -= (FLAG-1);}

            g->nindex[neig+1] += 1;
           
            if (!FLAG)
                soma += 1;
        }
        g->nindex[i+1] += soma;
    }

    // Adicionar os antihubs
    int contador = 0;

    for (long int i=0;i<nodes;i++)

        if (i == antihubs[contador]){
            contador ++;
            g->nindex[i+1] += (num_antihubs-1);
    }
     
    
    //Calcular offsets
    for (long int i=1;i<nodes+1;i++){

        g->nindex[i] = g->nindex[i-1] + g->nindex[i];

    }

}

void calculate_nlist(int nodes, int *kNN,int k, bool *flag_knn,ECLgraph *g,int *antihubs,int num_antihubs,long int *auxiliar_edges){

    long int k2 = k;

    // Adiciona os vizinhos paralelamente
    omp_set_num_threads(32);
    #pragma omp parallel for 
    for (long int i = 0; i < nodes; i++) {
        
        // Calcula o offset do ponto
        long int edge_offset = g->nindex[i];
        long int pos = edge_offset + auxiliar_edges[i];    

        for (long int j = 0; j < k2; j++) {
        
            // Pega o   ndice do vizinho  
            long int neig = kNN[i * k2 + j];

            g->nlist[pos] = neig;
            auxiliar_edges[i] += 1;

            pos += 1; 

        }
    }

    // Adiciona vizinhos que não são mútuos
    for (long int i = 0; i < nodes; i++) {
        

        for (long int j = 0; j < k2; j++) {
        
            // Pega o   indice do vizinho  
            long int neig = kNN[i * k2 + j];

            int FLAG = flag_knn[i*k2+j]; 

            // Deu problema
            if (!FLAG){

                //Calcula Propriedades de NEIG em NList
                long int neig_edge_offset = g->nindex[neig];


                long int neig_pos = neig_edge_offset + auxiliar_edges[neig];
                // Adicionando o idx i na lista do neig
                auxiliar_edges[neig] += 1;
                g->nlist[neig_pos] = i;
            }
        }
    }

    //Adiciona os antihubs
    for (long int i=0;i<num_antihubs;i++){

        int current = antihubs[i];
        long int pos_begin = g->nindex[current];
        long int offset = auxiliar_edges[current];

        for (long int j=0;j<num_antihubs;j++){
            int neig = antihubs[j];

            if (neig != current){
                g->nlist[pos_begin+offset] = neig;
                offset++;
            }
        }
    }

}


void calculate_coreDistance_antihubs(ECLgraph *g,long int *auxiliar_edges,int *antihubs,long int num_antihubs){


    // Read vector txtx
    const std::string path_to_data_binary = "/nndescent/GPU_HDBSCAN/data/vectors.fvecs";
    long int data_size2, data_dim2;
    float *vectors_data;

    FileTool::ReadBinaryAntihubs(path_to_data_binary,&vectors_data, &data_size2, &data_dim2,antihubs,num_antihubs);    	

    for (long int i=0;i<num_antihubs;i++){

        int idx_a = antihubs[i];
        long int pos_begin = g->nindex[idx_a] + auxiliar_edges[idx_a];
        for (long int j=i;j<num_antihubs-1;j++){
            int idx_b = antihubs[j+1];
            
            //Calcula distancia euclidiana
            float euclidean_distance = calculate_euclidean_distance(vectors_data,i,j+1,data_dim2);

            if (g->eweight[pos_begin + j] < euclidean_distance){
                g->eweight[pos_begin+j] = euclidean_distance;

                long int pos_begin2 = g->nindex[idx_b] + auxiliar_edges[idx_b];
                g->eweight[pos_begin2 + i] = euclidean_distance;
            }


        }
    }
}



ECLgraph buildECLgraph(int nodes, long int edges,int *kNN, float *distances,int k,long int mpts, int *antihubs, long int num_antihubs,int mst_gpu)
{


    long int numValues = nodes;
    ECLgraph g;


    g.nodes = nodes;


   /*Isso significa, que o nó 0 está conectado com Y-x NÓS,
   O nó 1 está conetado com z-y nós, e etc...*/
    hipMallocManaged(&g.nindex,(size_t)(g.nodes + 1) * sizeof(g.nindex[0])); // nindex[0] = X, nindex[1] = y, nindex[2] = z
    int gridSize = (g.nodes + 1 + blockSize - 1) / blockSize;
    initializeVectorCounts_<<<gridSize,blockSize>>>(g.nindex,0,g.nodes+1); // Aqui usar GPU
    Check();
    hipMemPrefetchAsync(g.nindex,(size_t)(g.nodes + 1) * sizeof(g.nindex[0]),hipCpuDeviceId);

    bool *flag_knn = (bool*)malloc(numValues*k * sizeof(bool));

    calculate_nindex(nodes, kNN, flag_knn,&g,antihubs,num_antihubs);

    // Nesse pontos os nós já estão calculados, agora precisamos inserir as arestas. Essa parte será bem demorada.
    long int *auxiliar_edges;

    hipMallocManaged(&auxiliar_edges,(size_t)(g.nodes) * sizeof(long int)); // nindex[0] = X, nindex[1] = y, nindex[2] = z
    gridSize = (g.nodes + 1  + blockSize - 1) / blockSize;
    initializeVectorCounts_<<<gridSize,blockSize>>>(auxiliar_edges,0,g.nodes); //Aqui usar GPU
    avoid_pageFault(auxiliar_edges,g.nodes,true);
    Check();


    hipMallocManaged(&g.nlist,(size_t)(g.nindex[nodes]) * sizeof(int));
    g.edges = g.nindex[nodes];
    calculate_nlist(nodes, kNN,k, flag_knn,&g,antihubs,num_antihubs,auxiliar_edges);


    hipMallocManaged(&g.eweight,(size_t)g.edges * sizeof(g.eweight[0]));

	int *aux_nodes;
    hipMallocManaged(&aux_nodes,(size_t)g.edges * sizeof(g.nlist[0]));
    createNodeList(aux_nodes,&g) ; //Aqui usar GPU
    Check();

         
    long int elementsPerGPU[numGPUs];
    calculateElements(elementsPerGPU,numGPUs,numValues); 

    float *coreDistances;
    hipMallocManaged(&coreDistances,(size_t)(numValues) * sizeof(float)); 
    calculateCoreDistance(distances,coreDistances,elementsPerGPU,k,mpts); //Aqui usa GPU
    Check();


    hipFree(kNN);
    kNN = NULL;

    if(mst_gpu != 1){hipFree(distances); distances = NULL; }


    calculateMutualReachabilityDistance(g.eweight,coreDistances,aux_nodes,g.nlist,g.edges);  //Aqui usa GPU
    Check();

    // Read vector txtx
    calculate_coreDistance_antihubs(&g,auxiliar_edges,antihubs,num_antihubs);

  return g;   
}


int* calculate_degrees(int *kNN,long int vectorSize,int shards_num,long int numValues){

    hipMemPrefetchAsync(kNN,(size_t)vectorSize * sizeof(int), hipCpuDeviceId);
 
    // Calcula a quantidade de elementos por GPU
    long int elementsPerGPU[shards_num];
    calculateElements(elementsPerGPU,shards_num,vectorSize);

    // Realiza a contagem de graus para todos os vértices
    int *finalCounts; // Contagens finais após a combinação das GPUs
    hipMallocManaged(&finalCounts,(size_t)numValues * sizeof(int));

    
    int gridSize = (numValues + blockSize - 1) / blockSize;
    // Inicializa o vetor
    initializeVectorCounts<<<gridSize,blockSize>>>(finalCounts,0,numValues);
    Check();        

    // Conta os graus de cada vértice
    countDegrees(finalCounts,kNN,numGPUs,elementsPerGPU,numValues);


    return finalCounts;
}


int* calculate_finalAntihubs(Vertex *vertexes,int *kNN,int* finalCounts,int* antihubs,long int numValues,int countsTreshold,
                            int pos_threshold, int value_threshold,long int k){

    // Pega os índices dos pontos que são iguais ao threshold
    int *treshold_idx;
	hipMallocManaged(&treshold_idx,(size_t)countsTreshold * sizeof(int));
    get_IndexThreshold(finalCounts,treshold_idx,value_threshold,numValues);

    avoid_pageFault(treshold_idx,countsTreshold);

    // Calcula quantos elementos serão processados por cada GPU
    long int indexesPerGPU[numGPUs];
    calculateElements(indexesPerGPU,numGPUs,countsTreshold);

    // Calculata os scores dos empates
    Untie_hub *unties = new Untie_hub[countsTreshold];
    calculateUntieScore(unties,indexesPerGPU,kNN,treshold_idx,finalCounts,k);

    // Pega quantos empates temos na lista final
    int missing_ties = get_TiedVertexes(vertexes,pos_threshold,value_threshold);
    int not_ties = pos_threshold - missing_ties;

    std::partial_sort(unties, unties + missing_ties, unties + countsTreshold, compareVertexByScore);

    // Junta todos os antihubs em um vetor
    joinAntiHubs(antihubs,vertexes,not_ties,unties,missing_ties);

    delete unties;
    unties = NULL;

    return antihubs;

}

ECLgraph buildEnhancedKNNG(int *kNN, float *distances, int shards_num, long int numValues,long int k,long int mpts ,int mst_gpu){

    long int vectorSize = numValues*k;

    int *finalCounts = calculate_degrees(kNN,vectorSize,shards_num,numValues);

    Vertex *vertexes;
	hipMallocManaged(&vertexes,(size_t)numValues * sizeof(Vertex));
    int gridSize = (numValues + blockSize - 1) / blockSize;    
    initializeVertex<<<gridSize,blockSize>>>(vertexes,finalCounts,numValues);
    Check();

    // Pegar os threshold  + Ordenação Parcial + Pegar o valor do threshold 
    int pos_threshold = get_NumThreshold(numValues);
    std::partial_sort(vertexes, vertexes + pos_threshold, vertexes + numValues, compareVertexByDegree); 
    int value_threshold = vertexes[pos_threshold-1].grau;
    printf("A posicao do threshold eh: %d e o valor eh: %d\n",pos_threshold-1,value_threshold);


    avoid_pageFault(vertexes,numValues);

    // Encontrar quantos valores são iguais ao threshold
    long int elementsPerGPU_[numGPUs];
    calculateElements(elementsPerGPU_,numGPUs,numValues);

    // Encontra quantos valores são iguais ao threshold
    int countsTreshold = countThreshold_(elementsPerGPU_,vertexes,value_threshold);

    int *antihubs;
    antihubs = new int[pos_threshold];

     if (countsTreshold > 1){
   
        calculate_finalAntihubs(vertexes,kNN,finalCounts,antihubs,numValues,countsTreshold,
                            pos_threshold, value_threshold,k);
     }

    else{
        // Bota os não empatados
        for(int i=0;i< pos_threshold;i++)
            antihubs[i] = vertexes[i].index;
    }

    // Ordena pelo índice para inserir na MST
    std::sort(antihubs,antihubs+pos_threshold);


    // Libera a galera
    hipFree(finalCounts);
    hipFree(vertexes);
    finalCounts = NULL;
    vertexes = NULL;


    ECLgraph g;
    
    g = buildECLgraph(numValues, vectorSize,kNN, distances,k,mpts, antihubs, pos_threshold,mst_gpu);

    return g;
}
