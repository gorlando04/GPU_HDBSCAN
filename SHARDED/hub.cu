#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <algorithm>
#include <time.h>


const int numGPUs = 3;
const long int numValues = 1000000; // Número de valores possíveis no vetor
const long int vectorSize = 32000000;// Tamanho do vetor
const int blockSize = 256;
const int k = 32;

struct Vertex {

    int index;
    int grau;
};

struct Untie_hub
{
    int index;
    int score=0;
};



__global__ void ShardVector(int *vec_dev,int *vectors,long int off_set_size,long int end,int offset=0){


    long int tid = (blockIdx.x * blockSize) + threadIdx.x;

    if (tid < end){
    vec_dev[tid] = vectors[tid + off_set_size*offset];
    }
}


// Função do kernel CUDA para calcular a contagem de valores em um vetor
__global__ void countValues(int *data, int *counts, long int size) {


    //int tid = blockIdx.x * blockDim.x + threadIdx.x;
    long int tid = (blockIdx.x * blockSize) + threadIdx.x;
    if (tid < size) {
        long int value = data[tid];
        atomicAdd(&counts[value], 1);
    }
}


__global__ void ShardVertex(Vertex *vec_dev,Vertex *vectors,long int off_set_size,long int end,int offset=0){


    long int tid = (blockIdx.x * blockSize) + threadIdx.x;

    if (tid < end){
    vec_dev[tid].grau = vectors[tid + off_set_size*offset].grau;

    }
}


// Função do kernel CUDA para calcular a contagem de valores em um vetor
__global__ void countTreshold(Vertex *data, int *counts, long int size,int comparision,int gpu_id) {


    //int tid = blockIdx.x * blockDim.x + threadIdx.x;
    long int tid = (blockIdx.x * blockSize) + threadIdx.x;
    if (tid < size) {
        long int value = data[tid].grau;

        if (value == comparision)
            atomicAdd(&counts[value], 1);
    }
}

__global__ void calculateScore(int *vectors,int *treshold_idx, Untie_hub *vertex , int *degrees ,long int size,int k,int offset){

    //Idx que iremos calcular o score
    long int tid = (blockIdx.x * blockSize) + threadIdx.x;

    if (tid < size){
        int idx = treshold_idx[tid+ (offset) ];
        vertex[tid].index = idx;

        for (int j=0;j<k;j++){
            int neig = vectors[tid+j];
            vertex[tid].score += degrees[neig];
        }
    }

}

// Custom comparator to sort MyStruct based on the 'value' field
bool compareVertexByDegree(const Vertex &a, const Vertex &b) {
    return a.grau < b.grau;
}


// Custom comparator to sort MyStruct based on the 'value' field
bool compareVertexByScore(const Untie_hub &a, const Untie_hub &b) {
    return a.score < b.score;
}



int main() {
    int shards_num = 9;
    int *h_data;  // Vetor na CPU
    int *d_data[shards_num], *d_counts[shards_num];  // Vetores na GPU
    int *h_counts[shards_num]; // Contagens na CPU para cada GPU
    int *finalCounts; // Contagens finais após a combinação das GPUs

    // Aloca memória para o vetor na CPU
	hipMallocManaged(&h_data,(size_t)vectorSize * sizeof(int));
  
    // Aqui da pra colocar um perfectch

    for (int i=0;i<numGPUs;i++){
        hipSetDevice(i);
        hipMemPrefetchAsync(h_data,(size_t)vectorSize * sizeof(int),i);
    }


    auto cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        printf("%s ",hipGetErrorString(cuda_status));
        exit(-1);
    }

// Gera o vetor de teste
  for (long int i = 0; i < vectorSize; i++) {
        h_data[i] = i / k;//rand() % numValues;
    }

    printf("Comecand a busca\n");

    clock_t t; 
    t = clock(); 


    // Inicializa contagens na CPU para cada GPU
    for (int i = 0; i < shards_num; i++) {
        h_counts[i] = new int[numValues];
        memset(h_counts[i], 0, numValues * sizeof(int));
    }

    long int elementsPerGPU[shards_num];

    //Define o tamanho de cada shard
    for (int i=0;i<shards_num;i++){

        if (i != (shards_num)-1){ elementsPerGPU[i] = vectorSize / (shards_num);}

        else{
            elementsPerGPU[i] = vectorSize;

            for (int j=0;j<i;j++)
                elementsPerGPU[i] -= elementsPerGPU[j];
            

        }
    }

// Realiza a contagem de graus para todos os vértices

 int iters = shards_num / numGPUs;

    for (int s=0;s < iters;s++){

	for (int i = 0; i < numGPUs; i++) {

        int idx = i + (s*numGPUs);
        hipSetDevice(i);

        // Aloca memória para o vetor e contagens na GPU
        hipMalloc(&d_data[idx],elementsPerGPU[idx] * sizeof(int)); // Vetor de valores
        // Aloca memória para o vetor e contagens na GPU
        hipMalloc(&d_counts[idx], numValues * sizeof(int)); // Vetor de frequências


        // Configura a grade de threads
        long int numBlocks = ( elementsPerGPU[idx]/ blockSize) +1;
        

        ShardVector<<< numBlocks,blockSize>>>(d_data[idx],h_data,elementsPerGPU[0],elementsPerGPU[idx], idx);

        
        //hipMemcpy(h_data2[idx], d_data[idx], elementsPerGPU[idx] * sizeof(int), hipMemcpyDeviceToHost);
        countValues<<<numBlocks, blockSize>>>(d_data[idx], d_counts[idx], elementsPerGPU[idx]);

        
        }

        //Libera a memória

        for (int i = 0; i < numGPUs; i++) {
            int idx = i + (s*numGPUs);
            hipSetDevice(i);

            hipDeviceSynchronize();

            hipMemcpy(h_counts[idx], d_counts[idx], numValues * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(d_data[idx]);
            hipFree(d_counts[idx]);  

        }    

        cuda_status = hipGetLastError();
        if (cuda_status != hipSuccess) {
            printf("%s hehehehehe",hipGetErrorString(cuda_status));
            exit(-1);
        }
    
	printf("Iter %d finalizada.\n",s);
    }






    
 // Combina as contagens de todas as GPUs na CPU
    finalCounts = new int[numValues];

    memset(finalCounts, 0, numValues * sizeof(int));

    for (int i = 0; i < shards_num; i++) {
        for (int j = 0; j < numValues; j++) {
            finalCounts[j] += h_counts[i][j];
        }
    }   


    /*for (long int i=0;i<numValues;i++){
	if (finalCounts[i] != 30){ printf("Resultado errado"); return 0;}
        printf("%ld = %d\n",i,finalCounts[i]);
    }*/

    // Atribui o vetor de contagens a uma struct que contém os índices

    Vertex *vertexes;

    //vertexes = new Vertex[numValues];
    // Aloca memória para o vetor na CPU
	hipMallocManaged(&vertexes,(size_t)numValues * sizeof(Vertex));
  
      for (int i=0;i<numGPUs;i++){
        hipSetDevice(i);
        hipMemPrefetchAsync(vertexes,(size_t)numValues * sizeof(Vertex),i);
    }

    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        printf("%s ",hipGetErrorString(cuda_status));
        exit(-1);
    }

    for (int i=0;i<numValues;i++){
        vertexes[i].grau = finalCounts[i];
        vertexes[i].index = i;
    }

    //Libera a memória


    // Ordenar o vetor de structs pelo valor do grau.

    // Pegar os threshold
    int pos_threshold = sqrt(numValues) ;

    std::partial_sort(vertexes, vertexes + pos_threshold, vertexes + numValues, compareVertexByDegree);


    // Pegar o valor do threshold 
    int value_threshold = vertexes[pos_threshold].grau;

    // Pega quantos empates temos na lista final

    int missing = 0;
    for (int i=pos_threshold-1;i > 0; i--){
        if (vertexes[i].grau  != value_threshold){
            missing = pos_threshold - i ;
            break;
        }
    }

    printf("A posicao do threshold eh: %d e o valor eh: %d\n",pos_threshold,value_threshold);

    
    // Encontrar quantos valores são iguais ao threshold
    Vertex *degree_count[numGPUs];  // Vetores na GPU

    long int elementsPerGPU_[numGPUs];

    //Define o tamanho de cada shard
    for (int i=0;i<numGPUs;i++){

        if (i != (numGPUs)-1){ elementsPerGPU_[i] = numValues / (numGPUs);}

        else{
            elementsPerGPU_[i] = numValues;

            for (int j=0;j<i;j++)
                elementsPerGPU_[i] -= elementsPerGPU_[j];
            

        }
    }

    int *gpu_count[numGPUs];

    int *cpu_counts[numGPUs]; // Contagens na CPU para cada GPU
    int countsTreshold = 0; // Contagens finais após a combinação das GPUs

    // Inicializa contagens na CPU para cada GPU
    for (int i = 0; i < numGPUs; i++) {
        cpu_counts[i] = new int[numGPUs];
        memset(cpu_counts[i], 0, numGPUs * sizeof(int));
    }


    printf("Buscando os vertices\n");

    for (int i = 0; i < numGPUs; i++) {

        hipSetDevice(i);

        // Aloca memória para o vetor e contagens na GPU
        hipMalloc(&degree_count[i],elementsPerGPU_[i] * sizeof(Vertex)); // Vetor de valores
        // Aloca memória para o vetor e contagens na GPU
        hipMalloc(&gpu_count[i],numGPUs*sizeof(int)); // Vetor de valores


        // Configura a grade de threads
        long int numBlocks = ( elementsPerGPU_[i]/ blockSize) +1;
        

        ShardVertex<<< numBlocks,blockSize>>>(degree_count[i],vertexes,elementsPerGPU_[0],elementsPerGPU_[i], i);

        
        countTreshold<<<numBlocks, blockSize>>>(degree_count[i], gpu_count[i], elementsPerGPU_[i],value_threshold,i);

           

        cuda_status = hipGetLastError();
        if (cuda_status != hipSuccess) {
            printf("%s hehehehehe",hipGetErrorString(cuda_status));
            exit(-1);
           }
    }



    for (int i = 0; i < numGPUs; i++) {
            hipSetDevice(i);

            hipDeviceSynchronize();

            hipMemcpy(cpu_counts[i], gpu_count[i], numGPUs * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(degree_count[i]);
            hipFree(gpu_count[i]);  

    }    


    for (int i=0;i<numGPUs;i++){
        // Quantidade de pontos que são iguais ao threshold
        countsTreshold += cpu_counts[i][0];
    }



    int *treshold_idx;

    // Aloca memória para o vetor na CPU
	hipMallocManaged(&treshold_idx,(size_t)countsTreshold * sizeof(int));
  
    for (int i=0;i<numGPUs;i++){
        hipSetDevice(i);
        hipMemPrefetchAsync(treshold_idx,(size_t)countsTreshold * sizeof(int),i);
    }

    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        printf("%s ",hipGetErrorString(cuda_status));
        exit(-1);
    }

    // Prepara o vetor com os índices dos pontos que são iguais ao threshold
    

    int control = 0;


    // Pega os índices iguais ao threshold.
    for (int i=0;i<numValues;i++){

        if (finalCounts[i] == value_threshold){
            treshold_idx[control] = i;
            control++;
        }
    }

    long int indexesPerGPU[numGPUs];

    //Define o tamanho de cada shard
    for (int i=0;i<numGPUs;i++){

        if (i != (numGPUs)-1){ indexesPerGPU[i] = countsTreshold/ (numGPUs);}

        else{
            indexesPerGPU[i] = countsTreshold;

            for (int j=0;j<i;j++)
                indexesPerGPU[i] -= indexesPerGPU[j];
            

        }
    }



    Untie_hub *unties_gpus[numGPUs];

    Untie_hub *unties_cpus[numGPUs];

    

    int *degree_counts_gpu[numGPUs];


    for (int i = 0; i < numGPUs; i++) {

        hipSetDevice(i);

        // Aloca memória para o vetor e contagens na GPU
        hipMalloc(&unties_gpus[i],indexesPerGPU[i] * sizeof(Untie_hub)); // Vetor de valores

        hipMalloc(&degree_counts_gpu[i], numValues * sizeof(int)); // Vetor de valores

        hipMemcpy(degree_counts_gpu[i], finalCounts, numValues * sizeof(int), hipMemcpyHostToDevice);


        // Configura a grade de threads
        long int numBlocks = ( indexesPerGPU[i]/ blockSize) +1;
        
        calculateScore<<<numBlocks,blockSize>>>(h_data,treshold_idx,unties_gpus[i],degree_counts_gpu[i],indexesPerGPU[i],k,i*indexesPerGPU[0]);

           

        cuda_status = hipGetLastError();
        if (cuda_status != hipSuccess) {
            printf("%s hehehehehe",hipGetErrorString(cuda_status));
            exit(-1);
           }
    }

    // Juntando tudo em CPU

    for (int i = 0; i < numGPUs; i++) {
            hipSetDevice(i);

            hipDeviceSynchronize();

            unties_cpus[i] = new Untie_hub[indexesPerGPU[i]];
            hipMemcpy(unties_cpus[i], unties_gpus[i], indexesPerGPU[i] * sizeof(Untie_hub), hipMemcpyDeviceToHost);

            hipFree(unties_gpus[i]);
            hipFree(degree_counts_gpu[i]);  

        cuda_status = hipGetLastError();
        if (cuda_status != hipSuccess) {
            printf("%s hehehehehe",hipGetErrorString(cuda_status));
            exit(-1);
           }

    }   

    Untie_hub *unties;
    unties = new Untie_hub[countsTreshold];


    // Junta em CPU
    for (int i=0;i<numGPUs;i++){

        for (int j=0;j<indexesPerGPU[i];j++){
            
            unties[ j + (i*indexesPerGPU[0])].index = unties_cpus[i][j].index;
            unties[ j + (i*indexesPerGPU[0])].score = unties_cpus[i][j].score;
        }
    }



    std::partial_sort(unties, unties + missing, unties + countsTreshold, compareVertexByScore);

    t = clock() - t; 
    double time_taken = ((double)t)/CLOCKS_PER_SEC; // in seconds 

    printf("Tempo total: %.3lf\n",time_taken);



    FILE *pFile;



    pFile=fopen("Time_1M_32.txt", "a");

    if(pFile==NULL) {
        perror("Error opening file.");
    }
else {

        fprintf(pFile, "%lf \n", time_taken);
    }

fclose(pFile);

    return 0;
}
